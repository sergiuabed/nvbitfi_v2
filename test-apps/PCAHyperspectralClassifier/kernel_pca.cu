#include "hip/hip_runtime.h"
/***
This software preprocess data using Principal Component Analysis ( PCA ) exploiting CUDA.
Modified by Gianluca De Lucia ( gianluca.delucia.94@gmail.com ) and Diego Romano ( diego.romano@cnr.it )
based on GPU_GSPCA code by Nathaniel Merrill.
***/

#include "kernel_pca.h"
#include <iostream>


KernelPCA::KernelPCA() : K(-1)
{
        // initialize cublas
        status = cublasInit();

        if(status != HIPBLAS_STATUS_SUCCESS)
        {
                std::runtime_error( "! CUBLAS initialization error\n");
        }
}



KernelPCA::KernelPCA(int num_pcs) : K(num_pcs)
{
        // initialize cublas
        status = cublasInit();

        if(status != HIPBLAS_STATUS_SUCCESS)
        {
                std::runtime_error( "! CUBLAS initialization error\n");
        }
}




KernelPCA::~KernelPCA()
{
	
        // shutdown
        status = cublasShutdown(); 
        if(status != HIPBLAS_STATUS_SUCCESS) 
        { 
                std::runtime_error( "! cublas shutdown error\n"); 
        } 


}

void KernelPCA::fit_transform(int M, int N, float *R, bool verbose, float* imgT)
{


	// maximum number of iterations
	int J = 10000;

	// max error
	float er = 1.0e-7;

        // if no K specified, or K > min(M, N)
        int K_;
        K_ = min(M, N);
        if (K == -1 || K > K_) K = K_;

	int n, j, k;

	// transfer the host matrix R to device matrix dR
	float *dR = 0;
	status = cublasAlloc(M*N, sizeof(dR[0]), (void**)&dR);

	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		std::runtime_error( "! cuda memory allocation error (dR)\n");
	}

	status = hipblasSetMatrix(M, N, sizeof(R[0]), R, M, dR, M);
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		std::runtime_error( "! cuda access error (write dR)\n");
	}

	// allocate device memory for T, P
	float *dT = 0;
	status = cublasAlloc(M*K, sizeof(dT[0]), (void**)&dT);
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		std::runtime_error( "! cuda memory allocation error (dT)\n");
	}

	float *dP = 0;
	status = cublasAlloc(N*K, sizeof(dP[0]), (void**)&dP);
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		std::runtime_error( "! cuda memory allocation error (dP)\n");
	}

	// allocate memory for eigenvalues
	float *L;
	L = (float*)malloc(K * sizeof(L[0]));;
	if(L == 0)
	{
		std::runtime_error( "! memory allocation error: T\n");
	}

	// mean center the data
	float *dU = 0;
	status = cublasAlloc(M, sizeof(dU[0]), (void**)&dU);
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		std::runtime_error( "! cuda memory allocation error (dU)\n");
	}

	hipblasScopy(M, &dR[0], 1, dU, 1);
	for(n=1; n<N; n++)
	{
		hipblasSaxpy (M, 1.0, &dR[n*M], 1, dU, 1);
	}

	for(n=0; n<N; n++)
	{
		hipblasSaxpy (M, -1.0/N, dU, 1, &dR[n*M], 1);
	}
	
	// GS-PCA
	float a;
	for(k=0; k<K; k++)
	{
		hipblasScopy (M, &dR[k*M], 1, &dT[k*M], 1);
		a = 0.0;
		for(j=0; j<J; j++)
		{
			hipblasSgemv ('t', M, N, 1.0, dR, M, &dT[k*M], 1, 0.0, &dP[k*N], 1);
			if(k>0)
			{
				hipblasSgemv ('t', N, k, 1.0, dP, N, &dP[k*N], 1, 0.0, dU, 1);
				hipblasSgemv ('n', N, k, -1.0, dP, N, dU, 1, 1.0, &dP[k*N], 1);
			}
			hipblasSscal (N, 1.0/hipblasSnrm2(N, &dP[k*N], 1), &dP[k*N], 1);
			hipblasSgemv ('n', M, N, 1.0, dR, M, &dP[k*N], 1, 0.0, &dT[k*M], 1);
			if(k>0)
			{
				hipblasSgemv ('t', M, k, 1.0, dT, M, &dT[k*M], 1, 0.0, dU, 1);
				hipblasSgemv ('n', M, k, -1.0, dT, M, dU, 1, 1.0, &dT[k*M], 1);
			}

			L[k] = hipblasSnrm2(M, &dT[k*M], 1);
			hipblasSscal(M, 1.0/L[k], &dT[k*M], 1);

			if(fabs(a - L[k]) < er*L[k]) break;
			
			a = L[k];
			
		}
			
		hipblasSger (M, N, - L[k], &dT[k*M], 1, &dP[k*N], 1, dR, M);
	
			
	
	}


	for(k=0; k<K; k++)
	{
		hipblasSscal(M, L[k], &dT[k*M], 1);
	}

	float *T;
        T = (float*)malloc(M*K * sizeof(T[0])); // user needs to free this outside this function

        if(T == 0)
        {
                std::runtime_error("! memory allocation error: T\n");
        }


	// transfer device dT to host T
	//hipblasGetMatrix (M, K, sizeof(dT[0]), dT, M, imgT, M);
        hipMemcpy(imgT,dT,sizeof(dT[0])*M*K,hipMemcpyDeviceToDevice);


	// clean up memory
	free(L);
	status = cublasFree(dP);
	status = cublasFree(dT);
	status = cublasFree(dR);
	status = cublasFree(dU);

}


void KernelPCA::set_n_components(int K_)
{
	K = K_;
}


int KernelPCA::get_n_components()
{
	return K;
}



